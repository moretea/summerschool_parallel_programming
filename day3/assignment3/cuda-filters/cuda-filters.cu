#include "hip/hip_runtime.h"
#include <math.h>
#include <iostream>
#include <stdlib.h>
#include "bmp_io.h"
#include "timer.h"

using namespace std;

#define HISTOGRAM_SIZE      256
#define CONTRAST_THRESHOLD   80
#define FILTER_WIDTH          3
#define FILTER_HEIGTH         3

float SOBEL_FILTER_X[FILTER_HEIGTH][FILTER_WIDTH] = { {-1,  0,  1}, {-2, 0, 2}, {-1, 0, 1} };
float SOBEL_FILTER_Y[FILTER_HEIGTH][FILTER_WIDTH] = { { 1,  2,  1}, { 0, 0, 0}, {-1,-2,-1} };

/* Utility function, use to do error checking.

   Use this function like this:

   checkCudaCall(hipMalloc((void **) &deviceRGB, imgS * sizeof(color_t)));

   And to check the result of a kernel invocation:

   checkCudaCall(hipGetLastError());
*/
static void checkCudaCall(hipError_t result) {
    if (result != hipSuccess) {
        cerr << "cuda error: " << hipGetErrorString(result) << endl;
        exit(1);
    }
}


// Utility function, do not parallelize
void save_histogram(unsigned int* histogram) {
    gray_image_t out = (gray_image_t)malloc(HISTOGRAM_SIZE*HISTOGRAM_SIZE*sizeof(float)); 

    unsigned max = 0;
    for (unsigned x=0; x<HISTOGRAM_SIZE; x++) {
        if(histogram[x] > max) max = histogram[x];
    }
    
    for (unsigned x=0; x<HISTOGRAM_SIZE; x++) {
        int val = HISTOGRAM_SIZE*histogram[x] / max;
        for (int y=0; y<val; y++) {
            out[y*HISTOGRAM_SIZE + x] = 0xFF;
        }
        for (unsigned y=val; y<HISTOGRAM_SIZE; y++) {
            out[y*HISTOGRAM_SIZE + x] = 0;
        }
    }
    
    write_GrayBMP("../results/cuda/histogram.bmp", out, HISTOGRAM_SIZE, HISTOGRAM_SIZE);
    free(out);
}

static fast_color_t *convert_to_fast(color_image_t org, int size) {
  fast_color_t *fast = (fast_color_t*) malloc(sizeof(fast));

  fast->R = (float*) malloc(sizeof(float) * size);
  fast->G = (float*) malloc(sizeof(float) * size);
  fast->B = (float*) malloc(sizeof(float) * size);

  for (int i = 0; i < size; i++) {
    fast->R[i] = org[i].R;
    fast->G[i] = org[i].G;
    fast->B[i] = org[i].B;
  }

  return fast;
}

static fast_color_t *malloc_and_copy_device_img(fast_color_t *host_img, int size) {
  fast_color_t *device_img;

  device_img = (fast_color_t*) malloc(sizeof(fast_color_t));

  checkCudaCall(hipMalloc( &(device_img->R), sizeof(float) * size));
  checkCudaCall(hipMalloc( &(device_img->G), sizeof(float) * size));
  checkCudaCall(hipMalloc( &(device_img->B), sizeof(float) * size));

  /* Copy */
  hipMemcpy(device_img->R, host_img->R, sizeof(float) * size, hipMemcpyHostToDevice);
  hipMemcpy(device_img->G, host_img->G, sizeof(float) * size, hipMemcpyHostToDevice);
  hipMemcpy(device_img->B, host_img->B, sizeof(float) * size, hipMemcpyHostToDevice);

  return device_img;
}

void free_device_img(fast_color_t *deviceImg) {
  hipFree(deviceImg->R);
  hipFree(deviceImg->G);
  hipFree(deviceImg->B);
  free(deviceImg);
}

__global__ void gray_kernel(float *R, float *G, float *B, float *gray) {
  unsigned int i = (blockDim.x * blockIdx.x + threadIdx.x) * 4;

  /* Load R */
  float r1 = R[i];
  float r2 = R[i+1];
  float r3 = R[i+2];
  float r4 = R[i+3];

  /* Load G */
  float g1 = G[i];
  float g2 = G[i+1];
  float g3 = G[i+2];
  float g4 = G[i+3];

  /* Load B */
  float b1 = B[i];
  float b2 = B[i+1];
  float b3 = B[i+2];
  float b4 = B[i+3];

  /* Calc */
  gray[i]     = (30 * r1 + 59 * g1 + 11 * b1) / 100;
  gray[i + 1] = (30 * r2 + 59 * g2 + 11 * b2) / 100;
  gray[i + 2] = (30 * r3 + 59 * g3 + 11 * b3) / 100;
  gray[i + 3] = (30 * r4 + 59 * g4 + 11 * b4) / 100;
}


hipEvent_t start, stop;
// Parallelize this function on the GPU
gray_image_t RGBtoGray(color_image_t RGB, int imgW, int imgH) {
    int imgS = imgW * imgH;
    gray_image_t gray;

    /* Convert image to a efficient float array*/
    fast_color_t *hostImg = convert_to_fast(RGB, imgS);

    /* Alloc & copy image to the GPU */
    fast_color_t *deviceImg = malloc_and_copy_device_img(hostImg, imgS);

    gray_image_t *device_out;
    checkCudaCall(hipMalloc((void **) &device_out, sizeof(float) * imgS));
    hipMemset(device_out,0, sizeof(float) * imgS);

    /* Compute gray */
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    gray_kernel<<< (imgS / 1024 / 4), 1024>>>(deviceImg->R, 
                                          deviceImg->G, 
                                          deviceImg->B, 
                                          (float*) device_out);
    hipEventRecord(stop, 0);

    /* Copy back */
    gray = (gray_image_t) malloc(imgS * sizeof(float));
    hipMemcpy(gray, device_out, imgS * sizeof(float), hipMemcpyDeviceToHost);

    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    
    cout << "kernel invocation for rgbtogray took " << (elapsedTime) << " milliseconds" << endl;

    /* Free image */
    free_device_img(deviceImg);
    return gray;
}

// Parallelize this function on the GPU
void histogram_1D(gray_image_t image, int imgSize, unsigned int *histogram) {
    for (int i=0; i<HISTOGRAM_SIZE; i++) {
        histogram[i]=0;
    }

    for (int i=0; i<imgSize; i++) {
        histogram[(int)image[i]]++;
    }
}

// Parallelize this function on the GPU
void contrast_1D(gray_image_t image, unsigned int *histogram, int imgSize) {    
    int i, min, max, diff;

    i=0;
    while (i<HISTOGRAM_SIZE && histogram[i] < CONTRAST_THRESHOLD) {
        i++;
    }
    min=i;

    i=HISTOGRAM_SIZE-1;
    while (i>min && histogram[i] < CONTRAST_THRESHOLD) { 
        i--;
    } 
    max=i;
    diff=max-min;
    
    for (i=0; i<imgSize; i++) { 
        if (image[i] < min) {
            image[i]=0;
        } else if (image[i] > max) {
            image[i]=255;
        } else {
            image[i] = 255 * (image[i]-min) / (float)diff;
        }
    }
}
                     

// Parallelize this function on the GPU
gray_image_t convolution2D(gray_image_t in, int imgW, int imgH, float filter[FILTER_HEIGTH][FILTER_WIDTH]) {
    int imgS = imgW * imgH;
    gray_image_t out;

    out = (gray_image_t) calloc(imgS, sizeof(float)); // calloc initializes the memory to 0.
    if (out == NULL) return NULL;

    // find center position of kernel (half of kernel size)
    int kCenterX = FILTER_WIDTH / 2;
    int kCenterY = FILTER_HEIGTH / 2;

    for(int y=0; y < imgH; y++) {
        for(int x=0; x < imgW; x++) {
        
            for(int m=0; m < FILTER_HEIGTH; ++m) {
                for(int n=0; n < FILTER_WIDTH; ++n) {
            
                    // index of input signal, used for checking boundary
                    int yy = y + (m - kCenterY);
                    int xx = x + (n - kCenterX);
            
                    // ignore input samples which are out of bound
                    if( yy >= 0 && yy < imgH && xx >= 0 && xx < imgW ) {
                        out[y*imgW+x] += in[yy*imgW+xx] * filter[m][n];
                    }
                }
            }
        }
    }
    
    return out;
}


// Parallelize this function on the GPU
// Combines in1 and in2, stores result in image in1 (overwriting the original).
void combineImages(gray_image_t in1, gray_image_t in2, int imgSize) {
    for (int i=0; i<imgSize; i++) {
        in1[i] = sqrt(in1[i]*in1[i]+in2[i]*in2[i]);
        if(in1[i] > 255.0f) {
            in1[i] = 255.0f;
        } 
    }
}


int main(int argc, char* argv[]) {
    int imgW, imgH;
    color_image_t image;
    unsigned int histogram[HISTOGRAM_SIZE];

    if (argc<2)   {
        cerr << "Not enough arguments! Bailing out..." << endl;
        return -1;
    }
    
    if ((image = read_BMP(argv[1], &imgW, &imgH)) == NULL) {
        cerr << "Cannot read BMP ... ?! " << endl;
        return 1;    
    }

    int imgSize = imgW * imgH;
    timer rgbToGrayTimer("rgb to gray");
    timer histogramTimer("histogram");
    timer contrastTimer("contrast");
    timer convolutionTimer("convolution");


    // Convert to grayscale image
    rgbToGrayTimer.start();
    gray_image_t gray = RGBtoGray(image, imgW, imgH);
    cout << "MEH" << endl;
    rgbToGrayTimer.stop();

    free(image);
    write_GrayBMP("../results/cuda/gray.bmp", gray, imgW, imgH);

    // Compute Histogram
    histogramTimer.start();
    histogram_1D(gray, imgSize, histogram);
    histogramTimer.stop();

    save_histogram(histogram);
    

    // Contrast Enhancement
    contrastTimer.start();
    contrast_1D(gray, histogram, imgSize);
    contrastTimer.stop();

    write_GrayBMP("../results/cuda/contrast.bmp", gray, imgW, imgH);
    

    // Convolution
    convolutionTimer.start();
    gray_image_t outX = convolution2D(gray, imgW, imgH, SOBEL_FILTER_X);
    gray_image_t outY = convolution2D(gray, imgW, imgH, SOBEL_FILTER_Y);
    combineImages(outX, outY, imgSize);
    convolutionTimer.stop();

    write_GrayBMP("../results/cuda/convolution.bmp", outX, imgW, imgH);
    
    cout << rgbToGrayTimer;
    cout << histogramTimer;
    cout << contrastTimer;
    cout << convolutionTimer;

    free(gray);
    free(outX);
    free(outY);
    
    return 0;
}
