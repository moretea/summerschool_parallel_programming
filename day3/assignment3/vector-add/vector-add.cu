#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include "timer.h"
#include <iostream>

using namespace std;

/* Utility function, use to do error checking.

   Use this function like this:

   checkCudaCall(hipMalloc((void **) &deviceRGB, imgS * sizeof(color_t)));

   And to check the result of a kernel invocation:

   checkCudaCall(hipGetLastError());
*/
static void checkCudaCall(hipError_t result) {
    if (result != hipSuccess) {
        cerr << "cuda error: " << hipGetErrorString(result) << endl;
        exit(1);
    }
}


__global__ void vectorAddKernel(float* deviceA, float* deviceB, float* deviceResult) {
    unsigned index = blockIdx.x * blockDim.x + threadIdx.x;
    deviceResult[index] = deviceA[index] + deviceB[index];
}


void vectorAddCuda(int n, float* a, float* b, float* result) {
    int threadBlockSize = 512;

    // allocate the vectors on the GPU
    float* deviceA = NULL;
    checkCudaCall(hipMalloc((void **) &deviceA, n * sizeof(float)));
    if (deviceA == NULL) {
        cout << "could not allocate memory!" << endl;
        return;
    }
    float* deviceB = NULL;
    checkCudaCall(hipMalloc((void **) &deviceB, n * sizeof(float)));
    if (deviceB == NULL) {
        checkCudaCall(hipFree(deviceA));
        cout << "could not allocate memory!" << endl;
        return;
    }
    float* deviceResult = NULL;
    checkCudaCall(hipMalloc((void **) &deviceResult, n * sizeof(float)));
    if (deviceResult == NULL) {
        checkCudaCall(hipFree(deviceA));
        checkCudaCall(hipFree(deviceB));
        cout << "could not allocate memory!" << endl;
        return;
    }

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // copy the original vectors to the GPU
    checkCudaCall(hipMemcpy(deviceA, a, n*sizeof(float), hipMemcpyHostToDevice));
    checkCudaCall(hipMemcpy(deviceB, b, n*sizeof(float), hipMemcpyHostToDevice));

    // execute kernel
    hipEventRecord(start, 0);
    vectorAddKernel<<<n/threadBlockSize, threadBlockSize>>>(deviceA, deviceB, deviceResult);
    hipEventRecord(stop, 0);

    // check whether the kernel invocation was successful
    checkCudaCall(hipGetLastError());

    // copy result back
    checkCudaCall(hipMemcpy(result, deviceResult, n * sizeof(float), hipMemcpyDeviceToHost));

    checkCudaCall(hipFree(deviceA));
    checkCudaCall(hipFree(deviceB));
    checkCudaCall(hipFree(deviceResult));

    // print the time the kernel invocation took, without the copies!
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    
    cout << "kernel invocation took " << (elapsedTime*1000) << " milliseconds" << endl;
}


int main(int argc, char* argv[]) {
    int n = 65536;
    timer vectorAddTimer("vector add timer");
    float* a = new float[n];
    float* b = new float[n];
    float* result = new float[n];

    // initialize the vectors.
    for(int i=0; i<n; i++) {
        a[i] = i;
        b[i] = i;
    }

    vectorAddTimer.start();
    vectorAddCuda(n, a, b, result);
    vectorAddTimer.stop();

    cout << vectorAddTimer;

    // verify the resuls
    for(int i=0; i<n; i++) {
        if(result[i] != 2*i) {
            cout << "error in results! Element " << i << " is " << result[i] << ", but should be " << (2*i) << endl;
            exit(1);
        }
    }
    cout << "results OK!" << endl;
            
    delete[] a;
    delete[] b;
    delete[] result;
    
    return 0;
}
